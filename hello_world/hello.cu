
#include <hip/hip_runtime.h>
#include <stdio.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


// __global__ functions, or "kernels", execute on the device
__global__ void hello_kernel(void)
{
    printf("Hello, world from the device!\n");
}

int main(void)
{
    // greet from the host
    printf("Hello, world from the host!\n");

    // launch a kernel with a single thread to greet from the device
    hello_kernel<<<1, 1>>>();

    gpuErrchk( hipPeekAtLastError() );

    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(cudaerr));

    return 0;
}